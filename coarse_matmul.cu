
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#define TILE_SIZE 32
#define COARSE_FACTOR 4

void matmul_cpu(const float* A, const float* B, float* C, int M, int N, int K) {
    for (int row = 0; row < M; ++row) {
        for (int col = 0; col < N; ++col) {
            float sum = 0.0f;
            for (int i = 0; i < K; ++i) {
                sum += A[row * K + i] * B[i * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

__global__ void mat_mul_tiled_coarse_kernel(
    float* matA, float* matB, float* matC, u_int32_t heightA, u_int32_t widthB, u_int32_t widthA
){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int colStart = blockDim.x * blockIdx.x * COARSE_FACTOR + threadIdx.x;

    __shared__ float A_s[TILE_SIZE][TILE_SIZE];
    __shared__ float B_s[TILE_SIZE][TILE_SIZE];
    
    const unsigned int NUM_TILES = (widthA + TILE_SIZE - 1)/TILE_SIZE;
    float sum[COARSE_FACTOR];
    for(unsigned int c=0; c < COARSE_FACTOR; ++c) {
        sum[c] = 0;
    }
    

    for (unsigned int tile_section = 0; tile_section < NUM_TILES; ++tile_section){
        const int tile_offset = TILE_SIZE*tile_section;
        const int innerCol = tile_offset + threadIdx.x;
        const int innerRow = tile_offset + threadIdx.y;

        if(row < heightA && innerCol < widthA){
            A_s[threadIdx.y][threadIdx.x] =  matA[widthA*row + innerCol];
        } else {
            A_s[threadIdx.y][threadIdx.x] = 0;
        }

        for(unsigned int c=0; c < COARSE_FACTOR; ++c) {
            unsigned int col_new = colStart + c*TILE_SIZE;
            if(innerRow < widthA && col_new < widthB){
                B_s[threadIdx.y][threadIdx.x] =  matB[widthB*innerRow + col_new];
            } else {
                B_s[threadIdx.y][threadIdx.x] = 0;
            }
    
            __syncthreads();
    
            for (unsigned int  inner_index=0; inner_index < TILE_SIZE; ++inner_index){
                float a = A_s[threadIdx.y][inner_index];
                float b = B_s[inner_index][threadIdx.x];
                sum[c] += a * b;
            }
    
            __syncthreads();
        }

    }

     for(unsigned int c=0; c < COARSE_FACTOR; ++c) {
        unsigned int col_new = colStart + c*TILE_SIZE;
        u_int32_t output_index = widthB * row + col_new;
        if (row < heightA and col_new < widthB){
            matC[output_index] = sum[c];
        }
    }

}

int main() {
    u_int32_t  matA_size, matB_size, matC_size, matA_bytes, matB_bytes, matC_bytes; 
    
    u_int32_t M = 1024, N = 1024, K = 1024;
    matA_size = M * K; // MxK input matrix
    matB_size = K * N; // KxN input matrix
    matC_size = M * N; // MxN output matrix

    matA_bytes = matA_size * sizeof(float);
    matB_bytes = matB_size * sizeof(float);
    matC_bytes = matC_size * sizeof(float);


    float *matA = new float[matA_size];
    float *matB = new float[matB_size];
    float *matC = new float[matC_size];
    float *matC_check = new float[matC_size];
    
    for (int i=0; i< matA_size; i++){
        matA[i] =  float(rand())/RAND_MAX;
    }
    
    for (int i=0; i< matB_size; i++){
        matB[i] =  float(rand())/RAND_MAX;
    }
    
    float *matA_d, *matB_d, *matC_d;

    hipMalloc((void**)&matA_d, matA_bytes);
    hipMalloc((void**)&matB_d, matB_bytes);
    hipMalloc((void**)&matC_d, matC_bytes);

    hipMemcpy(matA_d, matA, matA_bytes, hipMemcpyHostToDevice);
    hipMemcpy(matB_d, matB, matB_bytes, hipMemcpyHostToDevice);


    const dim3 blockSize(TILE_SIZE, TILE_SIZE);
    const dim3 gridSize(
        // (N + (blockSize.x) - 1) /blockSize.x/COARSE_FACTOR,
        static_cast<int>(std::ceil(static_cast<float>(N)/blockSize.x/COARSE_FACTOR)),
        static_cast<int>(std::ceil(static_cast<float>(M)/blockSize.y)),
        1
    );


    mat_mul_tiled_coarse_kernel<<<gridSize, blockSize>>>(
        matA_d, matB_d, matC_d, M, N, K
    );

    hipMemcpy(matC, matC_d, matC_bytes, hipMemcpyDeviceToHost);

    // matmul_cpu(matA, matB, matC_check, M, N, K);
    // float sum = 0;
    // for (int i = 0; i < matC_size; i++) {
    //     float a = matC_check[i] ;
    //     float b = matC[i];
    //     float diff = abs(a - b);
    //     if(diff > 1e-4){
    //          std::cout <<  a << " " << b << " " << diff << "at" << i << std::endl;
    //     }
    //     sum += diff;
    // }
    // std::cout << "sum: " << sum << std::endl;

    hipFree(matA_d);
    hipFree(matB_d);
    hipFree(matC_d);
    

    delete[] matA;
    delete[] matB;
    delete[] matC;
    delete[] matC_check;

    return 0;
}
