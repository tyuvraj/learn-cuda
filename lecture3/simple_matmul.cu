
#include <hip/hip_runtime.h>
#include <iostream>
/**
 * Simple rectangular matrix multiplication
 */
__global__ void mat_mul_kernel(
    float* matA, float* matB, float* matC, u_int32_t M, u_int32_t N, u_int32_t K
){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    float sum = 0;
    for (int i = 0; i < K; i++) {
        float a = matA[K*row + i];
        float b = matB[N*i + col];
        sum += a * b;
    }
    u_int32_t output_index = N * row + col;
    if (row < M and col < N){
        matC[output_index] = sum;
    }

}

int main() {
    u_int32_t  matA_size, matB_size, matC_size, matA_bytes, matB_bytes, matC_bytes; 
    
    u_int32_t M = 256, N = 512, K = 1024;
    matA_size = M * K; // MxK input matrix
    matB_size = K * N; // KxN input matrix
    matC_size = M * N; // MxN output matrix

    matA_bytes = matA_size * sizeof(float);
    matB_bytes = matB_size * sizeof(float);
    matC_bytes = matC_size * sizeof(float);


    float *matA = new float[matA_size];
    float *matB = new float[matB_size];
    float *matC = new float[matC_size];
    
    for (int i=0; i< matA_size; i++){
        matA[i] =  i; //float(rand())/RAND_MAX;
    }
    
    for (int i=0; i< matB_size; i++){
        matB[i] =  i; //float(rand())/RAND_MAX;
    }
    
    float *matA_d, *matB_d, *matC_d;

    hipMalloc((void**)&matA_d, matA_bytes);
    hipMalloc((void**)&matB_d, matB_bytes);
    hipMalloc((void**)&matC_d, matC_bytes);

    hipMemcpy(matA_d, matA, matA_bytes, hipMemcpyHostToDevice);
    hipMemcpy(matB_d, matB, matB_bytes, hipMemcpyHostToDevice);

    const dim3 blockSize(32, 32);
    const dim3 gridSize(
        (N + blockSize.x - 1) / blockSize.x,
        (M + blockSize.y - 1) / blockSize.y,
        1
    );

    mat_mul_kernel<<<gridSize, blockSize>>>(
        matA_d, matB_d, matC_d, M, N, K
    );

    hipMemcpy(matC, matC_d, matC_bytes, hipMemcpyDeviceToHost);

    hipFree(matA_d);
    hipFree(matB_d);
    hipFree(matC_d);
    

    delete[] matA;
    delete[] matB;
    delete[] matC;

    return 0;
}
